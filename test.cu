#include <hip/hip_runtime.h>
#include <stdio.h>


float * boats;

void checkError()
{
	hipError_t error = hipGetLastError();
  if(error != hipSuccess)
  {
    // print the CUDA error message and exit
    printf("CUDA error: %s\n", hipGetErrorString(error));
    exit(-1);
  }
}
__global__ void normalMAKernel(float * d)
{
    float a = 234.3;
    float b = 4672.323;
    float c = 392053.2345;

    d[0] = 1.0/sqrt(c);
}

__global__ void speedyMAKernel(float * d)
{
    float a = 234.3;
    float b = 4672.323;
    float c = 392053.2345;

    d[0] = __frsqrt_rn(c);
}

int main(int argc, char *args[])
{
    hipSetDevice(1);
    printf("I ran a\n");
    hipEvent_t normalStart,normalStop, fastStart,fastStop;
    hipEventCreate(&normalStart);
    hipEventCreate(&normalStop);
    hipEventCreate(&fastStart);
    hipEventCreate(&fastStop);

    hipError_t err = hipMalloc(&boats, 100 * sizeof(float));
    if ( err != hipSuccess ) return 0;

    hipEventRecord(normalStart);
    normalMAKernel<<<214748,1>>>(boats);
    speedyMAKernel<<<214748,1>>>(boats);
    checkError();
    hipDeviceSynchronize();
    hipEventRecord(normalStop);

    float milliseconds = 0;

    hipEventElapsedTime(&milliseconds,normalStart,normalStop);

    printf("It took %f",milliseconds);

}